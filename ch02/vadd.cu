#include "hip/hip_runtime.h"
#include "../utils.h"

__host__ __device__ float f(float a, float b) {
    return a + b;
}

void vecadd_cpu(float *x, float *y, float *z, unsigned int N) {
    for (unsigned int i = 0; i < N; i++) {
        z[i] = f(x[i], y[i]);
    }
}

__global__ void vecadd_kernel(float *x, float *y, float *z, unsigned int N) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N) {
        z[i] = f(x[i], y[i]);
    }
}

void vecadd_gpu(float *x, float *y, float *z, unsigned int N) {
    // Allocate gpu memory
    float *x_d, *y_d, *z_d;
    hipMalloc(&x_d, N*sizeof(float));
    hipMalloc(&y_d, N*sizeof(float));
    hipMalloc(&z_d, N*sizeof(float));

    // Copy data to gpu
    hipMemcpy(x_d, x, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(y_d, y, N*sizeof(float), hipMemcpyHostToDevice);

    // Launch kernel
    const unsigned int threads_per_block = 512;
    const unsigned int blocks_per_grid = (N + threads_per_block - 1) / threads_per_block;
    vecadd_kernel<<<blocks_per_grid, threads_per_block>>>(x_d, y_d, z_d, N);

    

    hipFree(x_d);
    hipFree(y_d);
    hipFree(z_d);
}

int main(int argc, char** argv) {
    hipDeviceSynchronize();

    unsigned int N = (argc > 1)?(atoi(argv[1])):(1<<25);
    float* x = (float*)malloc(N*sizeof(float));
    float* y = (float*)malloc(N*sizeof(float));
    float* z = (float*)malloc(N*sizeof(float));

    for (unsigned int i = 0; i < N; i++) {
        x[i] = rand();
        y[i] = rand();
    }

    HOST_TIC(0);
    vecadd_cpu(x, y, z, N);
    HOST_TOC(0);

    HOST_TIC(1);
    vecadd_gpu(x, y, z, N);
    hipError_t err = hipDeviceSynchronize();
    if (err != hipSuccess) {
        fprintf(stderr, "CUDA error: %s\n", hipGetErrorString(err));
    }
    HOST_TOC(1);
    free(x);
    free(y);
    free(z);
}