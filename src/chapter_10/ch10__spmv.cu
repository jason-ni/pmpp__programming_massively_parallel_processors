#include "hip/hip_runtime.h"
/*
 * Programming Massively Parallel Processors - 3ed
 * Chapter 10
 * Presents the sparse matrix storage and manipulation techniques.
 * The "nvixnu__" libraries used here are available at https://gist.github.com/nvixnu.
 *
 *  Created on: 18/12/2020
 *  Author: Nvixnu
 */

#include <stdio.h>
#include <string.h>
#include "ch10__config.h"
#include "nvixnu__array_utils.h"
#include "nvixnu__populate_arrays_utils.h"
#include "nvixnu__error_utils.h"

typedef struct {
	int non_zeros;
	int rows;
	int cols;
	int largest_row_width;
} sparse_t;

typedef struct {
	double *data;
	int *col_idx;
	int *row_ptr;
} csr_t;

typedef struct {
	double *data;
	int *idx;
} ell_t;

__global__
void ch10__ell_spmv_kernel(double *m_data, int *m_col_index, const int length, const int num_rows, double *v, double *y) {
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row < num_rows) {
		float dot = 0;
		for (int i = 0; i < length; i++) {
			dot += m_data[row+i*num_rows] * v[m_col_index[row+i*num_rows]];
		}
		y[row] += dot;
	}
}

__global__
void ch10__csr_spmv_kernel(double *m_data, int *m_col_index, int *m_row_ptr, const int num_rows, double *v, double *y) {
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row < num_rows) {
		float dot = 0;
		int row_start = m_row_ptr[row];
		int row_end = m_row_ptr[row+1];
		for (int elem = row_start; elem < row_end; elem++) {
			dot += m_data[elem] * v[m_col_index[elem]];
		}
		y[row] += dot;
	}
}

void ch10__csr_spmv_device(csr_t h_csr, double *h_v, double *h_y, sparse_t dims, kernel_config_t config){
	double *d_y, *d_v;
	csr_t d_csr;

	const int block_dim = config.block_dim.x;
	const int grid_dim = ceil(dims.rows/(double)block_dim);

	CCE(hipMalloc(&d_csr.data, dims.non_zeros*sizeof(double)));
	CCE(hipMalloc(&d_y, dims.rows*sizeof(double)));
	CCE(hipMalloc(&d_v, dims.cols*sizeof(double)));
	CCE(hipMalloc(&d_csr.col_idx, dims.non_zeros*sizeof(int)));
	CCE(hipMalloc(&d_csr.row_ptr, (dims.rows+1)*sizeof(int)));

	CCE(hipMemcpy(d_csr.data, h_csr.data, dims.non_zeros*sizeof(double), hipMemcpyHostToDevice));
	CCE(hipMemcpy(d_v, h_v, dims.cols*sizeof(double), hipMemcpyHostToDevice));
	CCE(hipMemcpy(d_csr.col_idx, h_csr.col_idx, dims.non_zeros*sizeof(int), hipMemcpyHostToDevice));
	CCE(hipMemcpy(d_csr.row_ptr, h_csr.row_ptr, (dims.rows+1)*sizeof(int), hipMemcpyHostToDevice));

	DEVICE_TIC(0);
	ch10__csr_spmv_kernel<<<grid_dim, block_dim>>>(d_csr.data, d_csr.col_idx, d_csr.row_ptr, dims.rows, d_v, d_y);
	CCLE();
	DEVICE_TOC(0);

	CCE(hipMemcpy(h_y, d_y, dims.rows*sizeof(double), hipMemcpyDeviceToHost));

	CCE(hipFree(d_y));
	CCE(hipFree(d_v));
	CCE(hipFree(d_csr.data));
	CCE(hipFree(d_csr.col_idx));
	CCE(hipFree(d_csr.row_ptr));
}


void ch10__ell_spmv_device(ell_t h_ell, double *h_v, double *h_y, sparse_t dims, kernel_config_t config){
	double *d_y, *d_v;
	ell_t d_ell;

	const int block_dim = config.block_dim.x;
	const int grid_dim = ceil(dims.rows/(double)block_dim);

	CCE(hipMalloc(&d_ell.data, dims.largest_row_width*sizeof(double)));
	CCE(hipMalloc(&d_y, dims.rows*sizeof(double)));
	CCE(hipMalloc(&d_v, dims.cols*sizeof(double)));
	CCE(hipMalloc(&d_ell.idx, dims.largest_row_width*sizeof(int)));

	CCE(hipMemcpy(d_ell.data, h_ell.data, dims.largest_row_width*sizeof(double), hipMemcpyHostToDevice));
	CCE(hipMemcpy(d_v, h_v, dims.cols*sizeof(double), hipMemcpyHostToDevice));
	CCE(hipMemcpy(d_ell.idx, h_ell.idx, dims.largest_row_width*sizeof(int), hipMemcpyHostToDevice));


	DEVICE_TIC(0);
	if(!strcmp(config.kernel_version, CH10__SPMV_ELL)){
		ch10__ell_spmv_kernel<<<grid_dim, block_dim>>>(d_ell.data, d_ell.idx, dims.largest_row_width, dims.rows, d_v, d_y);
	}else{
		printf("\nINVALID KERNEL VERSION\n");
		exit(1);
	}
	DEVICE_TOC(0);


	CCE(hipMemcpy(h_y, d_y, dims.rows*sizeof(double), hipMemcpyDeviceToHost));

	CCE(hipFree(d_ell.data));
	CCE(hipFree(d_y));
	CCE(hipFree(d_v));
	CCE(hipFree(d_ell.idx));
}


void ch10__csr_spmv_host(double *m_data, int *m_col_index, int *m_row_ptr, const int num_rows, double *v, double *y){
	HOST_TIC(0);
	for (int row = 0; row < num_rows; row++) {
		float dot = 0;
		int row_start = m_row_ptr[row];
		int row_end = m_row_ptr[row+1];
		for (int elem = row_start; elem < row_end; elem++) {
			dot += m_data[elem] * v[m_col_index[elem]];
		}
		y[row] += dot;
	}
	HOST_TOC(0)
}

void csr2ell(csr_t csr, ell_t ell){

}

void ch10__spmv(env_e env, kernel_config_t config){
	csr_t csr;
	double *y, *v;
	sparse_t dims = {CH10__INPUT_ROWS, CH10__INPUT_COLS, CH10__INPUT_NON_ZERO_LENGTH, CH10__INPUT_LARGEST_NONZERO_ROW_WIDTH};

	csr.data = (double *)malloc(CH10__INPUT_NON_ZERO_LENGTH*sizeof(double));
	csr.col_idx = (int *)malloc(CH10__INPUT_NON_ZERO_LENGTH*sizeof(int));
	csr.row_ptr = (int *)malloc((CH10__INPUT_ROWS + 1)*sizeof(int));

	v = (double *)malloc(CH10__INPUT_COLS*sizeof(double));
	y = (double *)malloc(CH10__INPUT_ROWS*sizeof(double));

	nvixnu__populate_array_from_file(CH10__CSR_DATA_FILEPATH, "%lf", CH10__INPUT_NON_ZERO_LENGTH, sizeof(double), csr.data);
	nvixnu__populate_array_from_file(CH10__CSR_DATA_FILEPATH, "%d", CH10__INPUT_NON_ZERO_LENGTH, sizeof(int), csr.col_idx);
	nvixnu__populate_array_from_file(CH10__CSR_DATA_FILEPATH, "%d", CH10__INPUT_ROWS + 1, sizeof(int), csr.row_ptr);
	nvixnu__populate_array_from_file(CH10__VECTOR_FILEPATH, "%lf", CH10__INPUT_COLS, sizeof(double), v);

	if(env == Host){
		ch10__csr_spmv_host(csr.data, csr.col_idx, csr.row_ptr, CH10__INPUT_ROWS, v, y);
	}else{
		if(!strcmp(config.kernel_version, CH10__SPMV_CSR)){
			ch10__csr_spmv_device(csr, v, y, dims, config);
		}else{
			ell_t ell;

			ell.data = (double *)malloc(CH10__INPUT_LARGEST_NONZERO_ROW_WIDTH*sizeof(double));
			ell.idx = (int *)malloc(CH10__INPUT_LARGEST_NONZERO_ROW_WIDTH*sizeof(int));

			csr2ell(csr, ell);

			ch10__ell_spmv_device(ell, v, y, dims, config);

			free(ell.data);
			free(ell.idx);
		}

	}

	printf("Last %d values:\n", PRINT_LENGTH);
	nvixnu__array_map(y + CH10__INPUT_ROWS - PRINT_LENGTH, sizeof(double), PRINT_LENGTH, nvixnu__print_item_double);

	free(csr.data);
	free(csr.col_idx);
	free(csr.row_ptr);
	free(y);
	free(v);

	return;
}
