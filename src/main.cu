#include "hip/hip_runtime.h"
/*
 ============================================================================
 Name        : main.cu
 Author      : Nvixnu
 Version     :
 Copyright   : 
 Description : Programming massively Parallel Processors with CUDA - 3ed.
 ============================================================================
 */

#include <stdio.h>
#include "nvixnu__cuda_devices_props.h"
#include "chapter_2/ch2__config.h"
#include "chapter_3/ch3__config.h"
#include "chapter_4/ch4__config.h"
#include "chapter_5/ch5__config.h"
#include "chapter_7/ch7__config.h"
#include "chapter_8/ch8__config.h"
#include "utils.h"


static inline void chapter_2_menu(){
	printf("Chapter 2\n");
	printf("Running [vec_add] on Device with 256 threads per block...:\n");
	ch2__vec_add(Device, {.block_dim = {256,1,1}});

	printf("\nRunning [vec_add] on Device with 1024 threads per block...:\n");
	ch2__vec_add(Device, {.block_dim = {1024,1,1}});

	printf("\nRunning [vec_add] on Host...\n");
	ch2__vec_add(Host, {});
}


static inline void chapter_3_menu(){
	int option = -1;
	while(option != 0){
		printf("\nCHAPTER 3:\n");
		switch(option){
		case 1:
			printf("Running [color_to_grayscale] on Device with 256 threads per block...:\n");
			ch3__color_to_grayscale(Device, {.block_dim = {16,16,1}});

			printf("\nRunning [color_to_grayscale] on Device with 1024 threads per block...:\n");
			ch3__color_to_grayscale(Device, {.block_dim = {32,32,1}});

			printf("\nRunning [color_to_grayscale] on Host...\n");
			ch3__color_to_grayscale(Host, {});
			option = -1;
			break;
		case 2:
			printf("Running [blur] on Device with 256 threads per block...:\n");
			ch3__blur(Device, {.block_dim = {16,16,1}});

			printf("\nRunning [blur] on Device with 1024 threads per block...:\n");
			ch3__blur(Device, {.block_dim = {32,32,1}});

			printf("\nRunning [blur] on Host...\n");
			ch3__blur(Host, {});
			option = -1;
			break;
		default:
			printf("\t\t[1] - Color to grayscale\n");
			printf("\t\t[2] - Blur\n");
			printf("\nPress the number of the algorithm or zero to go back.\n");
			scanf("%d", &option);
			setbuf(stdin, NULL);
		}
	}
}

static inline void chapter_4_menu(){
	printf("Chapter 4\n");
	printf("Running [matrix_mul] on Device with 256 threads per block...:\n");
	ch4__matrix_mul(Device, {.block_dim = {16,16,1}, .kernel_version = CH4__MATRIX_MUL_KERNEL_NAIVE});

	printf("\nRunning [matrix_mul] on Device with 1024 threads per block...:\n");
	ch4__matrix_mul(Device, {.block_dim = {32,32,1}, .kernel_version = CH4__MATRIX_MUL_KERNEL_NAIVE});

	printf("\nRunning [matrix_mul_tiled] on Device with 256 threads per block...:\n");
	ch4__matrix_mul(Device, {.block_dim = {16,16,1}, .kernel_version = CH4__MATRIX_MUL_KERNEL_TILED});

	printf("\nRunning [matrix_mul_tiled] on Device with 1024 threads per block...:\n");
	ch4__matrix_mul(Device, {.block_dim = {32,32,1}, .kernel_version = CH4__MATRIX_MUL_KERNEL_TILED});

	printf("\nRunning [matrix_mul] on Host...\n");
	ch4__matrix_mul(Host, {});
}

static inline void chapter_5_menu(){
	printf("Chapter 5\n");
	printf("Running [ch5__sum_reduction] on Device with 256 threads per block...:\n");
	ch5__sum_reduction(Device, {.block_dim = {256,1,1}});

	printf("\nRunning [ch5__sum_reduction] on Device with 1024 threads per block...:\n");
	ch5__sum_reduction(Device, {.block_dim = {1024,1,1}});

	printf("\nRunning [ch5__sum_reduction] on Host...\n");
	ch5__sum_reduction(Host, {});
}

static inline void chapter_7_menu(){
	int option = -1;
	while(option != 0){
		printf("\nCHAPTER 7:\n");
		switch(option){
		case 1:
			printf("Running [ch7__1d_convolution] on Device with 256 threads per block...:\n");
			ch7__1d_convolution(Device, {.block_dim = {256,1,1}});

			printf("\nRunning [ch7__1d_convolution] on Device with 1024 threads per block...:\n");
			ch7__1d_convolution(Device, {.block_dim = {1024,1,1}});

			printf("\nRunning [ch7__1d_convolution] on Host...\n");
			ch7__1d_convolution(Host, {});
			option = -1;
			break;
		case 2:
			printf("Running [ch7__2d_convolution] on Device with 256 threads per block...:\n");
			ch7__2d_convolution(Device, {.block_dim = {16,16,1}});

			printf("\nRunning [ch7__2d_convolution] on Device with 1024 threads per block...:\n");
			ch7__2d_convolution(Device, {.block_dim = {32,32,1}});

			printf("\nRunning [ch7__2d_convolution] on Host...\n");
			ch7__2d_convolution(Host, {});
			option = -1;
			break;
		default:
			printf("\t\t[1] - 1D convolution\n");
			printf("\t\t[2] - 2D convolution\n");
			printf("\nPress the number of the algorithm or zero to go back.\n");
			scanf("%d", &option);
			setbuf(stdin, NULL);
		}
	}
}

static inline void chapter_8_menu(){
	int option = -1;
	//Gets the max length of shared memory to use as SECTION_SIZE of the 3-phase algorithm
	hipDeviceProp_t device_props =  nvixnu__get_cuda_device_props(0);
	const int memory_bound_section_size = device_props.sharedMemPerBlock;
	const int memory_bound_section_length = memory_bound_section_size/sizeof(double);
	const int thread_bound_section_length = device_props.maxThreadsDim[0];

	while(option != 0){
		printf("\nCHAPTER 8:\n");
		printf("CH8__ARRAY_LENGTH: %d\n", CH8__ARRAY_LENGTH);
		switch(option){
		case 1:

			printf("\nRunning [ch8__partial_prefix_sum Kogge-Stone] on Device with %d threads per block...:\n", thread_bound_section_length);
			ch8__partial_prefix_sum(Device, {
					.block_dim = {thread_bound_section_length,1,1},
					.kernel_version = CH8__PREFIX_SUM_KOGGE_STONE
			}, 0);

			printf("\nRunning [ch8__partial_prefix_sum Brent-Kung] on Device with %d threads per block...:\n", thread_bound_section_length);
			ch8__partial_prefix_sum(Device, {
					.block_dim = {thread_bound_section_length,1,1},
					.kernel_version = CH8__PREFIX_SUM_BRENT_KUNG
			}, 0);

			printf("\nRunning [ch8__partial_prefix_sum for Kogge-Stone/Brent-Kung comparison] on Host...\n");
			ch8__partial_prefix_sum(Host, {}, thread_bound_section_length);

			printf("\nRunning [ch8__partial_prefix_sum 3 phase Kogge-Stone] on Device with %d threads per block and section length equals to %d...:\n", thread_bound_section_length, memory_bound_section_length);
			ch8__partial_prefix_sum(Device, {
					.block_dim = {thread_bound_section_length,1,1},
					.kernel_version = CH8__PREFIX_SUM_3_PHASE_KOGGE_STONE,
					.shared_memory_size = memory_bound_section_size
			}, 0);


			printf("\nRunning [ch8__partial_prefix_sum for 3 phase Kogge-Stone comparison] on Host...\n");
			ch8__partial_prefix_sum(Host, {}, memory_bound_section_length);


			option = -1;
			break;
		case 2:

			printf("\nRunning [ch8__full_prefix_sum Hierarchical 3 phase Kogge-Stone] on Device with %d threads per block and section length equals to %d...:\n", thread_bound_section_length, memory_bound_section_length);
			ch8__full_prefix_sum(Device, {
					.block_dim = {thread_bound_section_length,1,1},
					.kernel_version = CH8__HIERARCHICAL_PREFIX_SUM_3_PHASE_KOGGE_STONE,
					.shared_memory_size = memory_bound_section_size
			});

			printf("\nRunning [ch8__full_prefix_sum] on Host...\n");
			ch8__full_prefix_sum(Host, {});

			option = -1;
			break;
		default:
			printf("\t\t[1] - Partial prefix sum (scan by block/section)\n");
			printf("\t\t[2] - Full prefix sum (scan on entire array)\n");
			printf("\nPress the number of the algorithm or zero to go back.\n");
			scanf("%d", &option);
			setbuf(stdin, NULL);
		}
	}
}



int main(void){

	int main = -1;

	while(main != 0){
		switch(main){
		case 2:
			chapter_2_menu();
			main = -1;
			break;
		case 3:
			chapter_3_menu();
			main = -1;
			break;
		case 4:
			chapter_4_menu();
			main = -1;
			break;
		case 5:
			chapter_5_menu();
			main = -1;
			break;
		case 7:
			chapter_7_menu();
			main = -1;
			break;
		case 8:
			chapter_8_menu();
			main = -1;
			break;
		default:
			printf("\nCHAPTERS:\n");
			printf("\t[Chapter 2] - Data parallel computing (vector addition)\n");
			printf("\t[Chapter 3] - Scalable parallel execution (Image Grayscale and Blur)\n");
			printf("\t[Chapter 4] - Memory and data locality (Matrix Multiplication)\n");
			printf("\t[Chapter 5] - Performance considerations (Array reduction)\n");
			printf("\t[Chapter 7] - Parallel patterns: convolution (1D and 2D convolution)\n");
			printf("\t[Chapter 8] - Parallel patterns: prefix sum (Sequantial, Kogge-Stone and Brent-Kung)\n");
			printf("\nPress the chapter number or zero to exit.\n");
			scanf("%d", &main);
			setbuf(stdin, NULL);
		}
	}




	return 0;
}

