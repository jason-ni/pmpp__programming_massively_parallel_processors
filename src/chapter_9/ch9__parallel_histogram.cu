/*
 * Programming Massively Parallel Processors - 3ed
 * Chapter 9
 * Presents the parallel histogram with the privatization and aggegation techniques.
 * The "nvixnu__" libraries used here are available at https://gist.github.com/nvixnu.
 *
 *  Created on: 18/12/2020
 *  Author: Nvixnu
 */

#include <stdio.h>
#include <string.h>
#include "ch9__config.h"
#include "nvixnu__array_utils.h"
#include "nvixnu__populate_arrays_utils.h"
#include "nvixnu__error_utils.h"

void ch9__parallel_histogram_device(double *h_input, double *h_output, const int length, kernel_config_t config){
	double *d_input, *d_output;

	CCE(hipMalloc(&d_input, length*sizeof(double)));
	CCE(hipMalloc(&d_output, length*sizeof(double)));

	CCE(hipMemcpy(d_input, h_input, length*sizeof(double), hipMemcpyHostToDevice));
	CCE(hipMemcpy(d_output, h_output, length*sizeof(double), hipMemcpyHostToDevice));

	DEVICE_TIC(0);
	if(!strcmp(config.kernel_version, "XXX")){

	}else{
		printf("\nINVALID KERNEL VERSION\n");
		exit(1);
	}
	DEVICE_TOC(0);

	CCE(hipMemcpy(h_output, d_output, length*sizeof(double), hipMemcpyDeviceToHost));

	CCE(hipFree(d_input));
	CCE(hipFree(d_output));
}

void ch9__parallel_histogram_host(double *input, double *output, const int length){
	HOST_TIC(0);
	HOST_TOC(0)
}

void ch9__parallel_histogram(env_e env, kernel_config_t config){
	double *input, *output;

	input = (double *)malloc(CH9__ARRAY_LENGTH*sizeof(double));
	output = (double *)calloc(CH9__ARRAY_LENGTH, sizeof(double));

	nvixnu__populate_array_from_file(CH9__FILEPATH, "%lf,", CH9__ARRAY_LENGTH, sizeof(double), input);

	if(env == Host){
		ch9__parallel_histogram_host(input, output, CH9__ARRAY_LENGTH);
	}else{
		ch9__parallel_histogram_device(input, output, CH9__ARRAY_LENGTH, config);
	}

	printf("Last %d values:\n", PRINT_LENGTH);
	nvixnu__array_map(output + CH9__ARRAY_LENGTH - PRINT_LENGTH, sizeof(double), PRINT_LENGTH, nvixnu__print_item_double);

	free(input);
	free(output);

	return;
}
