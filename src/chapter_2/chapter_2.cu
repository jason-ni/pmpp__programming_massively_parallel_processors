#include "hip/hip_runtime.h"
/*
 * Programming Massively Parallel Processors - 3ed
 * Chapter 2
 * In this chapter the vector addition and the error handlers functions are presented.
 * The "nvixnu__" libraries used here are available at https://gist.github.com/nvixnu.
 *
 *  Created on: 27/11/2020
 *  Author: Nvixnu
 */

#include "chapter_2.h"
#include <stdio.h>
#include <math.h>
#include <time.h>
#include "../utils.h"
#include "../datasets_info.h" //Credit card dataset info
#include "nvixnu__array_utils.h" //Map and print functions
#include "nvixnu__error_utils.h"
#include "nvixnu__populate_arrays_utils.h"
#include "nvixnu__axpy.h"


void ch2__vec_add_host(){
	// Pointers to host arrays
	double *x, *y;

	// Time handlers
	float duration;
	struct timespec start, stop;

	//Allocates the heap memory
	x = (double*)malloc(N*sizeof(double));
	y = (double*)malloc(N*sizeof(double));

	//Populates the arrays
	nvixnu__populate_multiple_arrays_from_file(FILEPATH, "", "%lf,", "", N, sizeof(double), 2, x, y);

	//Run the host function
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &start);
	nvixnu__axpy_host(1.0, x, y, N);
	clock_gettime(CLOCK_PROCESS_CPUTIME_ID, &stop);

	//Prints the host result and the elapsed time
	duration = HOST_DURATION_MS(start, stop);
	printf("\nHost elapsed time: %lf ms\n", duration);
	printf("Last %d values:\n", PRINT_SIZE);
	nvixnu__array_map(y + (N - PRINT_SIZE), sizeof(double), PRINT_SIZE, nvixnu__print_item_double);

	free(x);
	free(y);
}

void ch2__vec_add_device(const int block_dim){
	// Pointers to host arrays
	double *h_x, *h_y;
	// Pointers to device arrays
	double *d_x, *d_y;

	// Device time handlers
	float duration;
	hipEvent_t start, stop;
	CCE(hipEventCreate(&start));
	CCE(hipEventCreate(&stop));


	//Allocates the heap memory
	h_x = (double*)malloc(N*sizeof(double));
	h_y = (double*)malloc(N*sizeof(double));

	//Populates the arrays
	nvixnu__populate_multiple_arrays_from_file(FILEPATH, "", "%lf,", "", N, sizeof(double), 2, h_x, h_y);

	//Allocates the global memory
	CCE(hipMalloc(&d_x, N*sizeof(double)));
	CCE(hipMalloc(&d_y, N*sizeof(double)));

	//Copies the arrays to GPU
	CCE(hipMemcpy(d_x, h_x, N*sizeof(double), hipMemcpyHostToDevice));
	CCE(hipMemcpy(d_y, h_y, N*sizeof(double), hipMemcpyHostToDevice));

	//Launches the kernel
	CCE(hipEventRecord(start));
	nvixnu__axpy_kernel<<<ceil(N/(block_dim*1.0)), block_dim>>>(1.0, d_x, d_y, N);
	CCLE();
	CCE(hipEventRecord(stop));

	//Calculates the elapsed time
	CCE(hipEventSynchronize(stop));
	CCE(hipEventElapsedTime(&duration, start, stop));

	//Copies the result back to the heap
	CCE(hipMemcpy(h_y, d_y, N*sizeof(double), hipMemcpyDeviceToHost));

	//Prints the kernel result and the elapsed time
	printf("\nKernel elapsed time: %f ms\n", duration);
	printf("Last %d values:\n", PRINT_SIZE);
	nvixnu__array_map(h_y + (N - PRINT_SIZE), sizeof(double), PRINT_SIZE, nvixnu__print_item_double);


	CCE(hipFree(d_x));
	CCE(hipFree(d_y));
	free(h_x);
	free(h_y);
}
