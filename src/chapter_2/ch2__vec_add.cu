#include "hip/hip_runtime.h"
/*
 * Programming Massively Parallel Processors - 3ed
 * Chapter 2
 * In this chapter the vector addition and the error handlers functions are presented.
 * The "nvixnu__" libraries used here are available at https://gist.github.com/nvixnu.
 *
 *  Created on: 27/11/2020
 *  Author: Nvixnu
 */

#include <stdio.h>
#include <math.h>
#include <time.h>
#include "chapter_2.h"
#include "nvixnu__array_utils.h" //Map and print functions
#include "nvixnu__error_utils.h"
#include "nvixnu__populate_arrays_utils.h"
#include "nvixnu__axpy.h"


void ch2__vec_add_device(double *h_x, double *h_y, kernel_config_t config){
	// Pointers to device arrays
	double *d_x, *d_y;

	//Allocates the global memory
	CCE(hipMalloc(&d_x, N*sizeof(double)));
	CCE(hipMalloc(&d_y, N*sizeof(double)));

	//Copies the arrays to GPU
	CCE(hipMemcpy(d_x, h_x, N*sizeof(double), hipMemcpyHostToDevice));
	CCE(hipMemcpy(d_y, h_y, N*sizeof(double), hipMemcpyHostToDevice));

	DEVICE_TIC(0);
	nvixnu__axpy_kernel<<<ceil(N/(config.block_dim.x*1.0)), config.block_dim.x>>>(1.0, d_x, d_y, N);
	CCLE();
	DEVICE_TOC(0);

	//Copies the result back to the heap
	CCE(hipMemcpy(h_y, d_y, N*sizeof(double), hipMemcpyDeviceToHost));


	printf("Last %d values:\n", PRINT_SIZE);
	nvixnu__array_map(h_y + (N - PRINT_SIZE), sizeof(double), PRINT_SIZE, nvixnu__print_item_double);


	CCE(hipFree(d_x));
	CCE(hipFree(d_y));
}

void ch2__vec_add_host(double *x, double *y){
	HOST_TIC(0);
	nvixnu__axpy_host(1.0, x, y, N);
	HOST_TOC(0);

	printf("Last %d values:\n", PRINT_SIZE);
	nvixnu__array_map(y + (N - PRINT_SIZE), sizeof(double), PRINT_SIZE, nvixnu__print_item_double);
}

void ch2__vec_add(env_e env, kernel_config_t config){
	// Pointers to host arrays
	double *x, *y;

	//Allocates the heap memory
	x = (double*)malloc(N*sizeof(double));
	y = (double*)malloc(N*sizeof(double));

	//Populates the arrays
	nvixnu__populate_multiple_arrays_from_file(FILEPATH, "", "%lf,", "", N, sizeof(double), 2, x, y);

	if(env == Host){
		ch2__vec_add_host(x, y);
	}else{
		ch2__vec_add_device(x, y, config);
	}

	free(x);
	free(y);
}
