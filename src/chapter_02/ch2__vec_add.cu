#include "hip/hip_runtime.h"
/*
 * Programming Massively Parallel Processors - 3ed
 * Chapter 2
 * In this chapter the vector addition and the error handlers functions are presented.
 * The "nvixnu__" libraries used here are available at https://gist.github.com/nvixnu.
 *
 *  Created on: 27/11/2020
 *  Author: Nvixnu
 */

#include <stdio.h>
#include <math.h>
#include <time.h>
#include "ch2__config.h"
#include "nvixnu__array_utils.h" //Map and print functions
#include "nvixnu__error_utils.h"
#include "nvixnu__populate_arrays_utils.h"
#include "nvixnu__axpy.h"


void ch2__vec_add_device(double *h_x, double *h_y, const int length, kernel_config_t config){
	// Pointers to device arrays
	double *d_x, *d_y;

	//Allocates the global memory
	CCE(hipMalloc(&d_x, length*sizeof(double)));
	CCE(hipMalloc(&d_y, length*sizeof(double)));

	//Copies the arrays to GPU
	CCE(hipMemcpy(d_x, h_x, length*sizeof(double), hipMemcpyHostToDevice));
	CCE(hipMemcpy(d_y, h_y, length*sizeof(double), hipMemcpyHostToDevice));

	DEVICE_TIC(0);
	nvixnu__axpy_kernel<<<ceil(length/(double)config.block_dim.x), config.block_dim.x>>>(1.0, d_x, d_y, length);
	CCLE();
	DEVICE_TOC(0);

	//Copies the result back to the heap
	CCE(hipMemcpy(h_y, d_y, length*sizeof(double), hipMemcpyDeviceToHost));


	CCE(hipFree(d_x));
	CCE(hipFree(d_y));
}

void ch2__vec_add_host(double *x, double *y, const int length){
	HOST_TIC(0);
	nvixnu__axpy_host(1.0, x, y, length);
	HOST_TOC(0);
}

void ch2__vec_add(env_e env, kernel_config_t config){
	// Pointers to host arrays
	double *x, *y;

	//Allocates the heap memory
	x = (double*)malloc(CH2__ARRAY_LENGTH*sizeof(double));
	y = (double*)malloc(CH2__ARRAY_LENGTH*sizeof(double));

	//Populates the arrays
	nvixnu__populate_multiple_arrays_from_file(CH2__FILEPATH, "", "%lf,", "", CH2__ARRAY_LENGTH, sizeof(double), 2, x, y);

	if(env == Host){
		ch2__vec_add_host(x, y, CH2__ARRAY_LENGTH);
	}else{
		ch2__vec_add_device(x, y, CH2__ARRAY_LENGTH, config);
	}

	printf("Last %d values:\n", PRINT_LENGTH);
	nvixnu__array_map(y + (CH2__ARRAY_LENGTH - PRINT_LENGTH), sizeof(double), PRINT_LENGTH, nvixnu__print_item_double);

	free(x);
	free(y);
}
