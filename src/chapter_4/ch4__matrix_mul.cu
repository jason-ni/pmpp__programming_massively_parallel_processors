#include "hip/hip_runtime.h"
/*
 * chapter_4.cu
 *
 *  Created on: 27/11/2020
 *  Author: Nvixnu
 */

#include <stdio.h>
#include <math.h>
#include "chapter_4.h"
#include "nvixnu__populate_arrays_utils.h"
#include "nvixnu__array_utils.h"
#include "nvixnu__error_utils.h"
#include "nvixnu__gemm.h"


void ch4__matrix_mul_device(double *h_A, double *h_B, double *h_C, const int i_length, const int j_length, const int k_length, kernel_config_t config){
	double *d_A, *d_B, *d_C;

	CCE(hipMalloc(&d_A, A_LENGTH*sizeof(double)));
	CCE(hipMalloc(&d_B, B_LENGTH*sizeof(double)));
	CCE(hipMalloc(&d_C, C_LENGTH*sizeof(double)));


	CCE(hipMemcpy(d_A, h_A, A_LENGTH*sizeof(double), hipMemcpyHostToDevice));
	CCE(hipMemcpy(d_B, h_B, B_LENGTH*sizeof(double), hipMemcpyHostToDevice));
	CCE(hipMemcpy(d_C, h_C, C_LENGTH*sizeof(double), hipMemcpyHostToDevice));

	dim3 block_dim(config.block_dim.x, config.block_dim.y, 1);
	dim3 grid_dim(ceil(K_LENGTH/(double)config.block_dim.x), ceil(I_LENGTH/(double)config.block_dim.y), 1);

	DEVICE_TIC(0);
	if(config.kernel_version == MATRIX_MUL_KERNEL_NAIVE){
		nvixnu__gemm_kernel<<<grid_dim, block_dim>>>(d_A, d_B, d_C, I_LENGTH, J_LENGTH, K_LENGTH);
	}else{
		const int shared_memory_length = 2*config.block_dim.x*config.block_dim.y*sizeof(double);
		nvixnu__tiled_gemm_kernel<<<grid_dim, block_dim, shared_memory_length>>>(d_A, d_B, d_C, I_LENGTH, J_LENGTH, K_LENGTH, config.block_dim.x);
	}
	CCLE();
	DEVICE_TOC(0);

	CCE(hipMemcpy(h_C, d_C, C_LENGTH*sizeof(double), hipMemcpyDeviceToHost));


	CCE(hipFree(d_A));
	CCE(hipFree(d_B));
	CCE(hipFree(d_C));
}

void ch4__matrix_mul_host(double *A, double *B, double *C, const int i_length, const int j_length, const int k_length){
	HOST_TIC(0);
	nvixnu__gemm_host(A, B, C, i_length, j_length, k_length);
	HOST_TOC(0);
}

void ch4__matrix_mul(env_e env, kernel_config_t config){
	double *A, *B, *C;

	A = (double*)malloc(A_LENGTH*sizeof(double));
	B = (double*)malloc(B_LENGTH*sizeof(double));
	C = (double*)calloc(C_LENGTH, sizeof(double));

	nvixnu__populate_array_from_file(MATRIX_A_PATH, "%lf,", A_LENGTH, sizeof(double), A);
	nvixnu__populate_array_from_file(MATRIX_B_PATH, "%lf,", B_LENGTH, sizeof(double), B);

	if(env == Host){
		ch4__matrix_mul_host(A, B, C, I_LENGTH, J_LENGTH, K_LENGTH);
	}else{
		ch4__matrix_mul_device(A, B, C, I_LENGTH, J_LENGTH, K_LENGTH, config);
	}


	printf("Last %d values:\n", PRINT_LENGTH);
	nvixnu__array_map(C + (C_LENGTH - PRINT_LENGTH), sizeof(double), PRINT_LENGTH, nvixnu__print_item_double);

	free(A);
	free(B);
	free(C);

	return;
}

